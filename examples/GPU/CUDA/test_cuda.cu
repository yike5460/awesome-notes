#include <iostream>
#include <hip/hip_runtime.h>

__global__ void hello_cuda() {
    printf("Hello, CUDA!\n");
}

int main() {
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
